#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <chrono>

#include "shader_tools/GLSLProgram.h"
#include "shader_tools/GLSLShader.h"
#include "gui/gl_window.h"

// OpenGL
#include <GLFW/glfw3.h>

// Renderer
#include "renderer/camera.cuh"
#include "renderer/renderer.cuh"
#include "renderer/scene.cuh"
#include "content/model_loader.h"
#include "renderer/device_mesh_loader.cuh"
#include "renderer/device_random.cuh"
#include "renderer/autofocus.cuh"
#include "renderer/device_texture_loader.cuh"
#include "renderer/device_material_loader.cuh"

#if defined(RENDER_DEBUG)
#define DEBUG_ASSERT_SDL(x) {                                   \
                                if((x) == -1) {                 \
                                    std::cerr                   \
                                        << "SDL call failed: "  \
                                        << SDL_GetError()       \
                                    exit(1);                    \
                                }                               \
                            }
#else
#define DEBUG_ASSERT_SDL(x) (x)
#endif

// OpenGL
// GLuint VBO, VAO, EBO;


// CUDA <-> OpenGL interop
GLuint opengl_tex_cuda;

#define WIDTH   1024
#define HEIGHT  512

#if defined(RENDER_DEBUG)
#define DEBUG_ASSERT_SDL_PTR(x) {                                   \
                                    if(!(x)) {                      \
                                        std::cerr                   \
                                            << "SDL call failed: "  \
                                            << SDL_GetError()       \
                                        exit(1);                    \
                                    }                               \
                                }
#else
#define DEBUG_ASSERT_SDL_PTR(x)
#endif

__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}


void keyboard_func(GLFWwindow *window, int key, int scancode, int action, int mods) {}

void check_for_gl_errors() {
    while (true) {
        const GLenum err = glGetError();
        if (err == GL_NO_ERROR) {
            break;
        }

        std::cerr << "GL Error: " << gluErrorString(err) << std::endl;
    }
}

void create_gl_texture(GLuint *gl_tex, unsigned int size_x, unsigned int size_y) {
    glGenTextures(1, gl_tex);
    glBindTexture(GL_TEXTURE_2D, *gl_tex);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, size_x, size_y, 0, GL_RGBA, GL_FLOAT, NULL);

    check_for_gl_errors();
}

void init_glfw() {
    if (!glfwInit()) {
        std::cerr << "glfwInit failed!" << std::endl;
        exit(1);
    }
}

void init_gl_buffers() {
    create_gl_texture(&opengl_tex_cuda, WIDTH, HEIGHT);
    check_for_gl_errors();
}

void display(Camera *camera, Scene *scene, Renderer &renderer, GlWindow &window, RandomGeneratorPool *random,
             size_t sample) {
    renderer.render(camera, scene, random, WIDTH, HEIGHT, sample);
    glfwPollEvents();


    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, opengl_tex_cuda);


    window.draw();

    check_for_gl_errors();

    window.swap();
}

void print_cuda_device_info() {
    int device_count = 0;
    hipError_t error_id = hipGetDeviceCount(&device_count);

    std::cout << "Using the following CUDA device: " << std::endl;

    if (error_id != hipSuccess) {
        std::cerr << "hipGetDeviceCount returned " << (int) error_id << "\n" << hipGetErrorString(error_id)
                  << std::endl;
        exit(1);
    }

    if (device_count == 0) {
        std::cout << "There are no available devices that support CUDA" << std::endl;
        exit(1);
    }

    int device_id = 0;

    hipSetDevice(device_id);
    hipDeviceProp_t device_properties{};
    hipGetDeviceProperties(&device_properties, device_id);

    std::cout << "  Name: " << device_properties.name << "\n";

    int driver_version, runtime_version;
    hipDriverGetVersion(&driver_version);
    hipRuntimeGetVersion(&runtime_version);

    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driver_version / 1000,
           (driver_version % 100) / 10, runtime_version / 1000, (runtime_version % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n\n", device_properties.major,
           device_properties.minor);

}

std::vector<TriangleFace> faces_from_indices(const std::vector<int> &indices) {
    std::vector<TriangleFace> faces;
    for (int i = 0; i < indices.size(); i += 3) {
        faces.push_back({indices[i], indices[i + 1], indices[i + 2]});
    }

    return faces;
}

template<typename T, typename... Args>
T *create_device_type(Args &&... args) {
    T *object;
    hipMallocManaged(&object, sizeof(T));
    return new(object) T(std::forward<Args>(args)...);
}

void handle_input(GLFWwindow *window, Camera *camera, Scene *scene) {

    auto speed = 3.0f;
    if (glfwGetKey(window, GLFW_KEY_W)) {
        camera->set_position(camera->position() + camera->direction() * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_S)) {
        camera->set_position(camera->position() - camera->direction() * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_D)) {
        auto right = glm::cross(camera->up(), camera->direction());
        camera->set_position(camera->position() + right * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_A)) {
        auto right = glm::cross(camera->up(), camera->direction());
        camera->set_position(camera->position() - right * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_Z)) {
        camera->set_position(camera->position() + camera->up() * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_X)) {
        camera->set_position(camera->position() - camera->up() * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_SPACE)) {
        device_autofocus(camera, scene, WIDTH, HEIGHT);
    }
}

double cursor_x;
double cursor_y;
bool mouselook_active = false;
bool needs_autofocus = false;

void mouse_button_callback(GLFWwindow *window, int button, int action, int mods) {
    if (button == GLFW_MOUSE_BUTTON_LEFT) {
        if (action == GLFW_PRESS) {
            glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
            glfwGetCursorPos(window, &cursor_x, &cursor_y);
            mouselook_active = true;
        } else if (action == GLFW_RELEASE) {
            glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
            mouselook_active = false;
            needs_autofocus = true;
        }
    }
}

glm::vec3 get_forward(const glm::mat4x4 &mat) {
    auto inverted = glm::inverse(mat);
    auto forward = glm::normalize(inverted[2]);
    return glm::vec3(forward);
}

void set_camera_direction(Camera *camera, float yaw, float pitch) {
    auto xz_rotation = glm::rotate(yaw, glm::vec3(0, 1, 0));
    auto right_vector = glm::cross(get_forward(xz_rotation), glm::vec3(0, 1, 0));
    auto final_rotation = glm::rotate(xz_rotation, pitch, right_vector);
    camera->set_direction(get_forward(final_rotation));
}

int main() {
    init_glfw();

    GlWindow window{"Hello, world!", WIDTH, HEIGHT, keyboard_func};

    init_gl_buffers();

    print_cuda_device_info();

    Renderer rend{opengl_tex_cuda, WIDTH, HEIGHT};

    auto camera = create_device_type<Camera>();

    float rot = 1.45f;
    //auto camera_position = glm::vec3(glm::cos(rot) * 10.0, 0.0000, glm::sin(rot) * 10.0f);
    auto camera_position = glm::vec3(90.0, 100.0, 200.0);
    auto camera_direction = glm::normalize(glm::vec3(0.0, 0.0, -60.0f) - camera_position);
    camera->set_position(camera_position);
    camera->set_direction(camera_direction);
    camera->set_up(glm::vec3(0.0, 1.0, 0.0));
    camera->set_field_of_view(75.0 * (3.1415 / 180.0));
    camera->set_blur_radius(0.3); // (0.03);
    camera->set_focal_length(60.0);
    camera->set_shutter_speed(0.0);
    camera->set_resolution(glm::vec2(WIDTH, HEIGHT));
    camera->update();

    DeviceMeshLoader mesh_loader;

    hipDeviceSetLimit(hipLimitStackSize, 2048);

    glfwPollEvents();
    glfwPollEvents();

    auto red_diffuse = DeviceTextureLoader{}.load("/home/emil/textures/Plastic007_4K-JPG/color.jpg");

    DeviceTextureLoader texture_loader;
    DeviceMaterialLoader material_loader{texture_loader};

    auto wall_material = material_loader.load("/home/emil/textures/Bricks059_4K-JPG/");
    wall_material.set_uv_scale(glm::vec2(4.0f, 4.0f));
    auto wood_material = material_loader.load("/home/emil/textures/WoodFloor043_4K-JPG/");

    auto nvidia_texture = texture_loader.load("/home/emil/textures/nvidia/color.jpg");

    std::vector<SceneEntity> entities;

    auto floor_mesh = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    floor_mesh[0]->set_material(wood_material);

    auto wall_mesh = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    wall_mesh[0]->set_material(wall_material);

    auto wall_mesh2 = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    wall_mesh2[0]->set_material(wall_material);

    auto crate = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    crate[0]->material().set_diffuse_map(nvidia_texture);
    crate[0]->material().set_uv_scale(glm::vec2(-1.0f, 1.0f));
    entities.emplace_back(
            crate[0],
            WorldTransformBuilder()
                    .with_translation({-200, 10, 200})
                    .with_uniform_scale(1.5f)
                    .build()
    );

    auto light_mesh = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    light_mesh[0]->material().set_emission(glm::vec3(1.0, 1.0, 1.0));

    auto dragon = mesh_loader.load("/home/emil/models/stanford_dragon/dragon.obj");
    dragon[0]->set_material(wall_material);
    // dragon[0]->material().set_reflectivity(1.0f);

    // Dragon
    /*entities.emplace_back(
            dragon[0],
            WorldTransformBuilder()
            .with_translation({0.0, 0.0, -300})
                    .with_uniform_scale(20.0f)
                    .build()
    );*/

    // Ceiling Light
    /*entities.emplace_back(light_mesh[0],
                          WorldTransformBuilder()
                                  .with_translation({0.0, 960.0, 0.0})
                                  .with_scale({10.0, 0.1, 10.0})
                                  .build());*/

    entities.emplace_back(light_mesh[0],
                          WorldTransformBuilder()
                                  .with_translation({200.0, 300.0, 0.0})
                                  .with_uniform_scale(1.0)
                                  .build());


    // Mesh size is 96x96x96 cm, scaled to 960x9.6x960cm
    // Floor
    entities.emplace_back(
            floor_mesh[0],
            WorldTransformBuilder()
                    .with_translation({0.0, 0.0, 0.0})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );

    // Ceiling
    entities.emplace_back(
            wall_mesh[0],
            WorldTransformBuilder()
                    .with_translation({0.0, 960.0, 0.0})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );

    // Front wall
    entities.emplace_back(
            wall_mesh[0],
            WorldTransformBuilder()
                    .with_translation({0.0, 480.0, 480.0})
                    .with_rotation({glm::pi<float>() / 2.0f, 0.0, 0.0})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );

    // Back wall
    entities.emplace_back(
            wall_mesh2[0],
            WorldTransformBuilder()
                    .with_translation({0.0, 480.0, -480.0})
                    .with_rotation({glm::pi<float>() / 2.0f, 0.0, -glm::pi<float>() / 2.0f})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );

    // Left wall
    entities.emplace_back(
            wall_mesh[0],
            WorldTransformBuilder()
                    .with_translation({-480.0, 480.0, 0.0})
                    .with_rotation({0.0f, -glm::pi<float>() / 2.0f, glm::pi<float>() / 2.0f})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );

    // Right wall
    entities.emplace_back(
            wall_mesh[0],
            WorldTransformBuilder()
                    .with_translation({480.0, 480.0, 0.0})
                    .with_rotation({0.0, glm::pi<float>() / 2.0f, -glm::pi<float>() / 2.0f})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );


    Scene *scene;
    hipMallocManaged(&scene, sizeof(Scene));
    new(scene) Scene;
    scene->build(entities);

    std::cout << "Creating random states..." << std::flush;
    auto random = create_device_type<RandomGeneratorPool>(2048 * 256, 123);
    std::cout << "Done." << std::endl;
    double rotation = 0.0;
    double total_duration = 0.0f;
    double max_duration = 0.0f;
    int frame_counter = 0;
    size_t sample = 0;

    glfwSetInputMode(window.handle(), GLFW_RAW_MOUSE_MOTION, GLFW_TRUE);
    glfwSetMouseButtonCallback(window.handle(), mouse_button_callback);


    auto run = true;

    float yaw = 2.31;
    float pitch = 0.015f;

    set_camera_direction(camera, yaw, pitch);
    device_autofocus(camera, scene, WIDTH, HEIGHT);
    while (run && !window.should_close()) {
        handle_input(window.handle(), camera, scene);

        if (glfwGetKey(window.handle(), GLFW_KEY_ESCAPE)) {
            run = false;
        }

        if (glfwGetKey(window.handle(), GLFW_KEY_P)) {
            dragon[0]->material().set_reflectivity(0.0f);
            floor_mesh[0]->material().set_reflectivity(0.0f);
        }
        if (glfwGetKey(window.handle(), GLFW_KEY_O)) {
            dragon[0]->material().set_reflectivity(1.0f);
            floor_mesh[0]->material().set_reflectivity(0.3f);
        }

        if(glfwGetKey(window.handle(), GLFW_KEY_Y)) {
            wall_mesh2[0]->material().set_diffuse_map(red_diffuse);
            sample = 0;
        }

        if (mouselook_active) {
            set_camera_direction(camera, yaw, pitch);

            double current_cursor_x, current_cursor_y;
            glfwGetCursorPos(window.handle(), &current_cursor_x, &current_cursor_y);

            yaw = yaw - (current_cursor_x - cursor_x) * 0.01f;
            pitch = pitch + (current_cursor_y - cursor_y) * 0.01f;
            pitch = glm::clamp(pitch, -1.3f, 1.3f);
            /*float speed = 1.0f;
            camera_rotation.y += (current_cursor_x - cursor_x) / 100.0f;

            camera->set_direction(glm::vec3(glm::sin(camera_rotation.y), 0.0, glm::cos(camera_rotation.y)));*/
            cursor_x = current_cursor_x;
            cursor_y = current_cursor_y;
        }

        if(needs_autofocus) {
            needs_autofocus = false;
            device_autofocus(camera, scene, WIDTH, HEIGHT);
        }

        if (camera->needs_update()) {
            camera->update();
            sample = 0;
        }

        auto start = std::chrono::high_resolution_clock::now();
        display(camera, scene, rend, window, random, sample);
        ++sample;
        auto end = std::chrono::high_resolution_clock::now();
        auto frame_duration = std::chrono::duration<double, std::milli>(end - start);
        frame_counter++;
        if (frame_duration.count() > max_duration) {
            max_duration = frame_duration.count();
        }
        total_duration += frame_duration.count();
        std::cout << '\r' << "Frame time: " << frame_duration.count() << "ms\t\t Avg (10 frames): "
                  << (total_duration / frame_counter) << "ms\t\t Max: " << max_duration << "ms\tt Sample: " << sample
                  // << "\t\tCamera: " << camera->position().x << ", " << camera->position().y << ", " << camera->position().z << ", yaw " << yaw << " pitch " << pitch
                  << "                    "
                  << std::flush;

        if (frame_counter == 10) {
            frame_counter = 0;
            total_duration = 0;
        }
        rotation += frame_duration.count() * 0.0005;

        check_for_gl_errors();
    }

    hipFree(camera);

    // https://stackoverflow.com/questions/14446495/cmake-project-structure-with-unit-tests
    // https://bitbucket.org/EmilNorden/physicstracer/src/master/CMakeLists.txt
    return 0;
}
