#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <chrono>

#include "shader_tools/GLSLProgram.h"
#include "shader_tools/GLSLShader.h"
#include "gui/GlWindow.h"
#include "renderer/renderer.cuh"

// OpenGL
#include <GL/glew.h>
#include <GLFW/glfw3.h>

#if defined(RENDER_DEBUG)
#define DEBUG_ASSERT_SDL(x) {                                   \
                                if((x) == -1) {                 \
                                    std::cerr                   \
                                        << "SDL call failed: "  \
                                        << SDL_GetError()       \
                                    exit(1);                    \
                                }                               \
                            }
#else
#define DEBUG_ASSERT_SDL(x) (x)
#endif

// OpenGL
GLuint VBO, VAO, EBO;
GLSLShader drawtex_f; // GLSL fragment shader
GLSLShader drawtex_v; // GLSL fragment shader
GLSLProgram shdrawtex; // GLSLS program for textured draw

// CUDA <-> OpenGL interop
GLuint opengl_tex_cuda;

#define WIDTH   800
#define HEIGHT  600

#if defined(RENDER_DEBUG)
#define DEBUG_ASSERT_SDL_PTR(x) {                                   \
                                    if(!(x)) {                      \
                                        std::cerr                   \
                                            << "SDL call failed: "  \
                                            << SDL_GetError()       \
                                        exit(1);                    \
                                    }                               \
                                }
#else
#define DEBUG_ASSERT_SDL_PTR(x)
#endif
__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

static const char *glsl_drawtex_vertshader_src =
        "#version 330 core\n"
        "layout (location = 0) in vec3 position;\n"
        "layout (location = 1) in vec3 color;\n"
        "layout (location = 2) in vec2 texCoord;\n"
        "\n"
        "out vec3 ourColor;\n"
        "out vec2 ourTexCoord;\n"
        "\n"
        "void main()\n"
        "{\n"
        "	gl_Position = vec4(position, 1.0f);\n"
        "	ourColor = color;\n"
        "	ourTexCoord = texCoord;\n"
        "}\n";

static const char *glsl_drawtex_fragshader_src =
        "#version 330 core\n"
        "uniform usampler2D tex;\n"
        "in vec3 ourColor;\n"
        "in vec2 ourTexCoord;\n"
        "out vec4 color;\n"
        "void main()\n"
        "{\n"
        "   	vec4 c = texture(tex, ourTexCoord);\n"
        "   	color = c / 255.0;\n"
        "}\n";

// QUAD GEOMETRY
GLfloat vertices[] = {
        // Positions          // Colors           // Texture Coords
        1.0f, 1.0f, 0.5f, 1.0f, 0.0f, 0.0f, 1.0f, 1.0f,  // Top Right
        1.0f, -1.0f, 0.5f, 0.0f, 1.0f, 0.0f, 1.0f, 0.0f,  // Bottom Right
        -1.0f, -1.0f, 0.5f, 0.0f, 0.0f, 1.0f, 0.0f, 0.0f,  // Bottom Left
        -1.0f, 1.0f, 0.5f, 1.0f, 1.0f, 0.0f, 0.0f, 1.0f // Top Left
};
// you can also put positions, colors and coordinates in seperate VBO's
GLuint indices[] = {  // Note that we start from 0!
        0, 1, 3,  // First Triangle
        1, 2, 3   // Second Triangle
};

void check_for_gl_errors() {
    while(true) {
        const GLenum err = glGetError();
        if(err == GL_NO_ERROR) {
            break;
        }

        std::cerr << "GL Error: " << gluErrorString(err) << std::endl;
    }
}

void create_gl_texture(GLuint* gl_tex, unsigned int size_x, unsigned int size_y) {
    glGenTextures(1, gl_tex);
    glBindTexture(GL_TEXTURE_2D, *gl_tex);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8UI_EXT, size_x, size_y, 0, GL_RGBA_INTEGER_EXT, GL_UNSIGNED_BYTE, NULL);

    check_for_gl_errors();
}

void init_opengl() {
    glewExperimental = GL_TRUE; // need this to enforce core profile
    GLenum err = glewInit();
    glGetError();
    if(err != GLEW_OK) {
        std::cerr << "glewInit failed: " << glewGetErrorString(err) << std::endl;
        exit(1);
    }
    glViewport(0, 0, WIDTH, HEIGHT);
    check_for_gl_errors();
}

/*void keyboard_func(GLFWwindow* window, int key, int scancode, int action, int mods){
}*/

void init_glfw() {
    if(!glfwInit()) {
        std::cerr << "glfwInit failed!" << std::endl;
        exit(1);
    }
}

void init_gl_buffers() {
    create_gl_texture(&opengl_tex_cuda, WIDTH, HEIGHT);

    drawtex_v = GLSLShader("Textured draw vertex shader", glsl_drawtex_vertshader_src, GL_VERTEX_SHADER);
    drawtex_f = GLSLShader("Textured draw fragment shader", glsl_drawtex_fragshader_src, GL_FRAGMENT_SHADER);
    shdrawtex = GLSLProgram(&drawtex_v, &drawtex_f);
    shdrawtex.compile();
    check_for_gl_errors();
}

void display(Renderer& renderer, GlWindow& window, int frame) {
    renderer.render(WIDTH, HEIGHT);
    glfwPollEvents();
    glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);

    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, opengl_tex_cuda);

    shdrawtex.use();
    glUniform1i(glGetUniformLocation(shdrawtex.program, "tex"), 0);

    glBindVertexArray(VAO); // binding VAO automatically binds EBO
    glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
    glBindVertexArray(0); // unbind VAO

    check_for_gl_errors();

    window.swap();
}


int main() {
    init_glfw();
    GlWindow window{"Hello, world!", WIDTH, HEIGHT};

    init_opengl();

    init_gl_buffers();

    Renderer rend{opengl_tex_cuda, WIDTH, HEIGHT};

    // Generate buffers
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glGenBuffers(1, &EBO);

    // Buffer setup
    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

    // Position attribute (3 floats)
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(GLfloat), (GLvoid*)0);
    glEnableVertexAttribArray(0);
    // Color attribute (3 floats)
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(GLfloat), (GLvoid*)(3 * sizeof(GLfloat)));
    glEnableVertexAttribArray(1);
    // Texture attribute (2 floats)
    glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 8 * sizeof(GLfloat), (GLvoid*)(6 * sizeof(GLfloat)));
    glEnableVertexAttribArray(2);

    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // Note that this is allowed, the call to glVertexAttribPointer registered VBO as the currently bound
    // vertex buffer object so afterwards we can safely unbind
    glBindVertexArray(0);

    int frame = 0;
    while(!window.should_close()) {
        auto start = std::chrono::high_resolution_clock::now();
        display(rend, window, frame);
        //glfwWaitEvents();
        frame++;
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration<double>(end-start);
        std::cout << "Time to render 1 frame: " << duration.count() << std::endl;
        // std::cout << "FPS: " << (1000.0 / duration.count()) << std::endl;
    }

    int N = 1<<20;
    float *x, *y;

    // Allocate unified memory - accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for(int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for(int i = 0; i < N; i++) {
        maxError = std::max(maxError, std::abs(y[i]-3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    // https://stackoverflow.com/questions/14446495/cmake-project-structure-with-unit-tests
    // https://bitbucket.org/EmilNorden/physicstracer/src/master/CMakeLists.txt
    return 0;
}
