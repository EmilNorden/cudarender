#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <chrono>

#include "shader_tools/GLSLProgram.h"
#include "shader_tools/GLSLShader.h"
#include "gui/gl_window.h"

// OpenGL
#include <GLFW/glfw3.h>

// Renderer
#include "renderer/camera.cuh"
#include "renderer/renderer.cuh"
#include "renderer/scene.cuh"
#include "content/model_loader.h"
#include "renderer/device_mesh_loader.cuh"
#include "renderer/device_random.cuh"
#include "renderer/autofocus.cuh"
#include "renderer/device_texture_loader.cuh"

#if defined(RENDER_DEBUG)
#define DEBUG_ASSERT_SDL(x) {                                   \
                                if((x) == -1) {                 \
                                    std::cerr                   \
                                        << "SDL call failed: "  \
                                        << SDL_GetError()       \
                                    exit(1);                    \
                                }                               \
                            }
#else
#define DEBUG_ASSERT_SDL(x) (x)
#endif

// OpenGL
// GLuint VBO, VAO, EBO;
GLSLShader drawtex_f; // GLSL fragment shader
GLSLShader drawtex_v; // GLSL fragment shader
GLSLProgram shdrawtex; // GLSLS program for textured draw

// CUDA <-> OpenGL interop
GLuint opengl_tex_cuda;

#define WIDTH   1024
#define HEIGHT  512

#if defined(RENDER_DEBUG)
#define DEBUG_ASSERT_SDL_PTR(x) {                                   \
                                    if(!(x)) {                      \
                                        std::cerr                   \
                                            << "SDL call failed: "  \
                                            << SDL_GetError()       \
                                        exit(1);                    \
                                    }                               \
                                }
#else
#define DEBUG_ASSERT_SDL_PTR(x)
#endif

__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

static const char *glsl_drawtex_vertshader_src =
        "#version 330 core\n"
        "layout (location = 0) in vec3 position;\n"
        "layout (location = 1) in vec3 color;\n"
        "layout (location = 2) in vec2 texCoord;\n"
        "\n"
        "out vec3 ourColor;\n"
        "out vec2 ourTexCoord;\n"
        "\n"
        "void main()\n"
        "{\n"
        "	gl_Position = vec4(position, 1.0f);\n"
        "	ourColor = color;\n"
        "	ourTexCoord = texCoord;\n"
        "}\n";

static const char *glsl_drawtex_fragshader_src =
        "#version 330 core\n"
        "uniform sampler2D tex;\n"
        "in vec3 ourColor;\n"
        "in vec2 ourTexCoord;\n"
        "out vec4 color;\n"
        "void main()\n"
        "{\n"
        "   	vec4 c = texture(tex, ourTexCoord);\n"
        "   	color = c;\n"
        "}\n";

void keyboard_func(GLFWwindow *window, int key, int scancode, int action, int mods) {}

void check_for_gl_errors() {
    while (true) {
        const GLenum err = glGetError();
        if (err == GL_NO_ERROR) {
            break;
        }

        std::cerr << "GL Error: " << gluErrorString(err) << std::endl;
    }
}

void create_gl_texture(GLuint *gl_tex, unsigned int size_x, unsigned int size_y) {
    glGenTextures(1, gl_tex);
    glBindTexture(GL_TEXTURE_2D, *gl_tex);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, size_x, size_y, 0, GL_RGBA, GL_FLOAT, NULL);

    check_for_gl_errors();
}

void init_glfw() {
    if (!glfwInit()) {
        std::cerr << "glfwInit failed!" << std::endl;
        exit(1);
    }
}

void init_gl_buffers() {
    create_gl_texture(&opengl_tex_cuda, WIDTH, HEIGHT);

    drawtex_v = GLSLShader("Textured draw vertex shader", glsl_drawtex_vertshader_src, GL_VERTEX_SHADER);
    drawtex_f = GLSLShader("Textured draw fragment shader", glsl_drawtex_fragshader_src, GL_FRAGMENT_SHADER);
    shdrawtex = GLSLProgram(&drawtex_v, &drawtex_f);
    shdrawtex.compile();
    check_for_gl_errors();
}

void display(Camera *camera, Scene *scene, Renderer &renderer, GlWindow &window, RandomGeneratorPool *random,
             size_t sample) {
    renderer.render(camera, scene, random, WIDTH, HEIGHT, sample);
    glfwPollEvents();


    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, opengl_tex_cuda);

    shdrawtex.use();
    glUniform1i(glGetUniformLocation(shdrawtex.program, "tex"), 0);

    window.draw();

    check_for_gl_errors();

    window.swap();
}

void print_cuda_device_info() {
    int device_count = 0;
    hipError_t error_id = hipGetDeviceCount(&device_count);

    std::cout << "Using the following CUDA device: " << std::endl;

    if (error_id != hipSuccess) {
        std::cerr << "hipGetDeviceCount returned " << (int) error_id << "\n" << hipGetErrorString(error_id)
                  << std::endl;
        exit(1);
    }

    if (device_count == 0) {
        std::cout << "There are no available devices that support CUDA" << std::endl;
        exit(1);
    }

    int device_id = 0;

    hipSetDevice(device_id);
    hipDeviceProp_t device_properties{};
    hipGetDeviceProperties(&device_properties, device_id);

    std::cout << "  Name: " << device_properties.name << "\n";

    int driver_version, runtime_version;
    hipDriverGetVersion(&driver_version);
    hipRuntimeGetVersion(&runtime_version);

    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driver_version / 1000,
           (driver_version % 100) / 10, runtime_version / 1000, (runtime_version % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n\n", device_properties.major,
           device_properties.minor);

}

std::vector<TriangleFace> faces_from_indices(const std::vector<int> &indices) {
    std::vector<TriangleFace> faces;
    for (int i = 0; i < indices.size(); i += 3) {
        faces.push_back({indices[i], indices[i + 1], indices[i + 2]});
    }

    return faces;
}

template<typename T, typename... Args>
T *create_device_type(Args &&... args) {
    T *object;
    hipMallocManaged(&object, sizeof(T));
    return new(object) T(std::forward<Args>(args)...);
}

void handle_input(GLFWwindow *window, Camera *camera, Scene *scene) {

    auto speed = 3.0f;
    if (glfwGetKey(window, GLFW_KEY_W)) {
        camera->set_position(camera->position() + camera->direction() * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_S)) {
        camera->set_position(camera->position() - camera->direction() * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_D)) {
        auto right = glm::cross(camera->up(), camera->direction());
        camera->set_position(camera->position() + right * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_A)) {
        auto right = glm::cross(camera->up(), camera->direction());
        camera->set_position(camera->position() - right * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_Z)) {
        camera->set_position(camera->position() + camera->up() * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_X)) {
        camera->set_position(camera->position() - camera->up() * speed);
    }
    if (glfwGetKey(window, GLFW_KEY_SPACE)) {
        device_autofocus(camera, scene, WIDTH, HEIGHT);
    }
}

double cursor_x;
double cursor_y;
bool mouselook_active = false;
bool needs_autofocus = false;

void mouse_button_callback(GLFWwindow *window, int button, int action, int mods) {
    if (button == GLFW_MOUSE_BUTTON_LEFT) {
        if (action == GLFW_PRESS) {
            glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
            glfwGetCursorPos(window, &cursor_x, &cursor_y);
            mouselook_active = true;
        } else if (action == GLFW_RELEASE) {
            glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
            mouselook_active = false;
            needs_autofocus = true;
        }
    }
}

/*fn get_forward(mat: &glm::Mat4) -> glm::Vec3 {
let inverted = glm::inverse(mat);
let forward = glm::normalize(inverted[2]);
glm::vec3(forward.x, forward.y, forward.z)
}*/

glm::vec3 get_forward(const glm::mat4x4 &mat) {
    auto inverted = glm::inverse(mat);
    auto forward = glm::normalize(inverted[2]);
    return glm::vec3(forward);
}

void set_camera_direction(Camera *camera, float yaw, float pitch) {
    auto xz_rotation = glm::rotate(yaw, glm::vec3(0, 1, 0));
    auto right_vector = glm::cross(get_forward(xz_rotation), glm::vec3(0, 1, 0));
    auto final_rotation = glm::rotate(xz_rotation, pitch, right_vector);
    camera->set_direction(get_forward(final_rotation));
}

int main() {
    init_glfw();

    GlWindow window{"Hello, world!", WIDTH, HEIGHT, keyboard_func};

    init_gl_buffers();

    print_cuda_device_info();

    Renderer rend{opengl_tex_cuda, WIDTH, HEIGHT};

    auto camera = create_device_type<Camera>();

    float rot = 1.45f;
    //auto camera_position = glm::vec3(glm::cos(rot) * 10.0, 0.0000, glm::sin(rot) * 10.0f);
    auto camera_position = glm::vec3(90.0, 100.0, 200.0);
    auto camera_direction = glm::normalize(glm::vec3(0.0, 0.0, -60.0f) - camera_position);
    camera->set_position(camera_position);
    camera->set_direction(camera_direction);
    camera->set_up(glm::vec3(0.0, 1.0, 0.0));
    camera->set_field_of_view(75.0 * (3.1415 / 180.0));
    camera->set_blur_radius(0.3); // (0.03);
    camera->set_focal_length(60.0);
    camera->set_shutter_speed(0.0);
    camera->set_resolution(glm::vec2(WIDTH, HEIGHT));
    camera->update();

    DeviceMeshLoader mesh_loader;

    // auto house = mesh_loader.load("/home/emil/models/apple/apple.obj"); // 0.5 0.35 0.5
    // auto house = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    // auto suzanne = house[0];

    //std::vector<IndexedDeviceMesh> meshes;
    //meshes.push_back(suzanne);

    hipDeviceSetLimit(hipLimitStackSize, 2048);

    glfwPollEvents();
    auto wall = DeviceTextureLoader{}.load("/home/emil/textures/Bricks059_4K-JPG/color.jpg");
    auto wall_normal = DeviceTextureLoader{}.load("/home/emil/textures/Bricks059_4K-JPG/normal.jpg");
    auto wall_roughness = DeviceTextureLoader{}.load("/home/emil/textures/Bricks059_4K-JPG/Bricks059_4K_Roughness.jpg");
    glfwPollEvents();
    auto wood_diffuse = DeviceTextureLoader{}.load("/home/emil/textures/WoodFloor043_4K-JPG/color.jpg");
    auto wood_normal = DeviceTextureLoader{}.load("/home/emil/textures/WoodFloor043_4K-JPG/normal.jpg");

    auto nvidia_diffuse = DeviceTextureLoader{}.load("/home/emil/textures/nvidia/color.jpg");

    auto red_diffuse = DeviceTextureLoader{}.load("/home/emil/textures/Plastic007_4K-JPG/color.jpg");


    std::vector<SceneEntity> entities;

    auto floor_mesh = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    floor_mesh[0]->material().set_diffuse_map(wood_diffuse);
    floor_mesh[0]->material().set_uv_scale(glm::vec2(6.0f, 6.0f));
    floor_mesh[0]->material().set_reflectivity(0.3f);
    // floor_mesh[0]->material().set_normal_map(wood_normal);

    auto wall_mesh = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    wall_mesh[0]->material().set_diffuse_map(wall);
    wall_mesh[0]->material().set_roughness_map(wall_roughness);
    wall_mesh[0]->material().set_uv_scale(glm::vec2(4.0f, 4.0f));

    auto wall_mesh2 = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    wall_mesh2[0]->material().set_diffuse_map(wall);
    wall_mesh2[0]->material().set_uv_scale(glm::vec2(4.0f, 4.0f));
    //wall_mesh[0]->material().set_normal_map(wall_normal);

    auto crate = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    crate[0]->material().set_diffuse_map(nvidia_diffuse);
    crate[0]->material().set_uv_scale(glm::vec2(-1.0f, 1.0f));
    entities.emplace_back(
            crate[0],
            WorldTransformBuilder()
                    .with_translation({-200, 10, 200})
                    .with_uniform_scale(1.5f)
                    .build()
    );

    auto light_mesh = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    light_mesh[0]->material().set_emission(glm::vec3(1.0, 1.0, 1.0));

    auto dragon = mesh_loader.load("/home/emil/models/stanford_dragon/dragon.obj");
    dragon[0]->material().set_diffuse_map(wall);
    // dragon[0]->material().set_reflectivity(1.0f);

    // Dragon
    /*entities.emplace_back(
            dragon[0],
            WorldTransformBuilder()
            .with_translation({0.0, 0.0, -300})
                    .with_uniform_scale(20.0f)
                    .build()
    );*/

    // Ceiling Light
    /*entities.emplace_back(light_mesh[0],
                          WorldTransformBuilder()
                                  .with_translation({0.0, 960.0, 0.0})
                                  .with_scale({10.0, 0.1, 10.0})
                                  .build());*/

    entities.emplace_back(light_mesh[0],
                          WorldTransformBuilder()
                                  .with_translation({200.0, 300.0, 0.0})
                                  .with_uniform_scale(1.0)
                                  .build());


    // Mesh size is 96x96x96 cm, scaled to 960x9.6x960cm
    // Floor
    entities.emplace_back(
            floor_mesh[0],
            WorldTransformBuilder()
                    .with_translation({0.0, 0.0, 0.0})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );

    // Ceiling
    entities.emplace_back(
            wall_mesh[0],
            WorldTransformBuilder()
                    .with_translation({0.0, 960.0, 0.0})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );

    // Front wall
    entities.emplace_back(
            wall_mesh[0],
            WorldTransformBuilder()
                    .with_translation({0.0, 480.0, 480.0})
                    .with_rotation({glm::pi<float>() / 2.0f, 0.0, 0.0})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );

    // Back wall
    entities.emplace_back(
            wall_mesh2[0],
            WorldTransformBuilder()
                    .with_translation({0.0, 480.0, -480.0})
                    .with_rotation({glm::pi<float>() / 2.0f, 0.0, -glm::pi<float>() / 2.0f})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );

    // Left wall
    entities.emplace_back(
            wall_mesh[0],
            WorldTransformBuilder()
                    .with_translation({-480.0, 480.0, 0.0})
                    .with_rotation({0.0f, -glm::pi<float>() / 2.0f, glm::pi<float>() / 2.0f})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );

    // Right wall
    entities.emplace_back(
            wall_mesh[0],
            WorldTransformBuilder()
                    .with_translation({480.0, 480.0, 0.0})
                    .with_rotation({0.0, glm::pi<float>() / 2.0f, -glm::pi<float>() / 2.0f})
                    .with_scale({10.0, 0.1, 10.0})
                    .build()
    );


    Scene *scene;
    hipMallocManaged(&scene, sizeof(Scene));
    new(scene) Scene;
    scene->build(entities);

    std::cout << "Creating random states..." << std::flush;
    auto random = create_device_type<RandomGeneratorPool>(2048 * 256, 123);
    std::cout << "Done." << std::endl;
    double rotation = 0.0;
    double total_duration = 0.0f;
    double max_duration = 0.0f;
    int frame_counter = 0;
    size_t sample = 0;

    glfwSetInputMode(window.handle(), GLFW_RAW_MOUSE_MOTION, GLFW_TRUE);
    glfwSetMouseButtonCallback(window.handle(), mouse_button_callback);


    auto run = true;

    float yaw = 2.31;
    float pitch = 0.015f;

    set_camera_direction(camera, yaw, pitch);
    device_autofocus(camera, scene, WIDTH, HEIGHT);
    while (run && !window.should_close()) {
        handle_input(window.handle(), camera, scene);

        if (glfwGetKey(window.handle(), GLFW_KEY_ESCAPE)) {
            run = false;
        }

        if (glfwGetKey(window.handle(), GLFW_KEY_P)) {
            dragon[0]->material().set_reflectivity(0.0f);
            floor_mesh[0]->material().set_reflectivity(0.0f);
        }
        if (glfwGetKey(window.handle(), GLFW_KEY_O)) {
            dragon[0]->material().set_reflectivity(1.0f);
            floor_mesh[0]->material().set_reflectivity(0.3f);
        }

        if(glfwGetKey(window.handle(), GLFW_KEY_Y)) {
            wall_mesh2[0]->material().set_diffuse_map(red_diffuse);
            sample = 0;
        }

        if (mouselook_active) {
            set_camera_direction(camera, yaw, pitch);

            double current_cursor_x, current_cursor_y;
            glfwGetCursorPos(window.handle(), &current_cursor_x, &current_cursor_y);

            yaw = yaw - (current_cursor_x - cursor_x) * 0.01f;
            pitch = pitch + (current_cursor_y - cursor_y) * 0.01f;
            pitch = glm::clamp(pitch, -1.3f, 1.3f);
            /*float speed = 1.0f;
            camera_rotation.y += (current_cursor_x - cursor_x) / 100.0f;

            camera->set_direction(glm::vec3(glm::sin(camera_rotation.y), 0.0, glm::cos(camera_rotation.y)));*/
            cursor_x = current_cursor_x;
            cursor_y = current_cursor_y;
        }

        if(needs_autofocus) {
            needs_autofocus = false;
            device_autofocus(camera, scene, WIDTH, HEIGHT);
        }

        if (camera->needs_update()) {
            camera->update();
            sample = 0;
        }

        auto start = std::chrono::high_resolution_clock::now();
        display(camera, scene, rend, window, random, sample);
        ++sample;
        auto end = std::chrono::high_resolution_clock::now();
        auto frame_duration = std::chrono::duration<double, std::milli>(end - start);
        frame_counter++;
        if (frame_duration.count() > max_duration) {
            max_duration = frame_duration.count();
        }
        total_duration += frame_duration.count();
        std::cout << '\r' << "Frame time: " << frame_duration.count() << "ms\t\t Avg (10 frames): "
                  << (total_duration / frame_counter) << "ms\t\t Max: " << max_duration << "ms\tt Sample: " << sample
                  // << "\t\tCamera: " << camera->position().x << ", " << camera->position().y << ", " << camera->position().z << ", yaw " << yaw << " pitch " << pitch
                  << "                    "
                  << std::flush;

        if (frame_counter == 10) {
            frame_counter = 0;
            total_duration = 0;
        }
        rotation += frame_duration.count() * 0.0005;

        check_for_gl_errors();
    }

    hipFree(camera);

    // https://stackoverflow.com/questions/14446495/cmake-project-structure-with-unit-tests
    // https://bitbucket.org/EmilNorden/physicstracer/src/master/CMakeLists.txt
    return 0;
}
