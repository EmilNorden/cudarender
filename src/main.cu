#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <chrono>

#include "shader_tools/GLSLProgram.h"
#include "shader_tools/GLSLShader.h"
#include "gui/gl_window.h"

// OpenGL
#include <GLFW/glfw3.h>

// Renderer
#include "renderer/camera.cuh"
#include "renderer/renderer.cuh"
#include "renderer/scene.cuh"
#include "content/model_loader.h"
#include "renderer/device_mesh_loader.cuh"
#include "renderer/device_random.cuh"
#include "renderer/autofocus.cuh"
#include "renderer/device_texture_loader.cuh"

#if defined(RENDER_DEBUG)
#define DEBUG_ASSERT_SDL(x) {                                   \
                                if((x) == -1) {                 \
                                    std::cerr                   \
                                        << "SDL call failed: "  \
                                        << SDL_GetError()       \
                                    exit(1);                    \
                                }                               \
                            }
#else
#define DEBUG_ASSERT_SDL(x) (x)
#endif

// OpenGL
// GLuint VBO, VAO, EBO;
GLSLShader drawtex_f; // GLSL fragment shader
GLSLShader drawtex_v; // GLSL fragment shader
GLSLProgram shdrawtex; // GLSLS program for textured draw

// CUDA <-> OpenGL interop
GLuint opengl_tex_cuda;

#define WIDTH   1024
#define HEIGHT  512

#if defined(RENDER_DEBUG)
#define DEBUG_ASSERT_SDL_PTR(x) {                                   \
                                    if(!(x)) {                      \
                                        std::cerr                   \
                                            << "SDL call failed: "  \
                                            << SDL_GetError()       \
                                        exit(1);                    \
                                    }                               \
                                }
#else
#define DEBUG_ASSERT_SDL_PTR(x)
#endif

__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

static const char *glsl_drawtex_vertshader_src =
        "#version 330 core\n"
        "layout (location = 0) in vec3 position;\n"
        "layout (location = 1) in vec3 color;\n"
        "layout (location = 2) in vec2 texCoord;\n"
        "\n"
        "out vec3 ourColor;\n"
        "out vec2 ourTexCoord;\n"
        "\n"
        "void main()\n"
        "{\n"
        "	gl_Position = vec4(position, 1.0f);\n"
        "	ourColor = color;\n"
        "	ourTexCoord = texCoord;\n"
        "}\n";

static const char *glsl_drawtex_fragshader_src =
        "#version 330 core\n"
        "uniform sampler2D tex;\n"
        "in vec3 ourColor;\n"
        "in vec2 ourTexCoord;\n"
        "out vec4 color;\n"
        "void main()\n"
        "{\n"
        "   	vec4 c = texture(tex, ourTexCoord);\n"
        "   	color = c;\n"
        "}\n";
void keyboard_func(GLFWwindow* window, int key, int scancode, int action, int mods) {}
/*
Camera *camera;
Scene * scene;

void keyboard_func(GLFWwindow* window, int key, int scancode, int action, int mods){
    auto speed = 0.1f;
    if(key == GLFW_KEY_W) {
        camera->set_position(camera->position() + camera->direction() * speed);
    }
    else if(key == GLFW_KEY_S) {
        camera->set_position(camera->position() - camera->direction() * speed);
    }
    else if(key == GLFW_KEY_D) {
        auto right = glm::cross(camera->up(), camera->direction());
        camera->set_position(camera->position() + right * speed);
    }
    else if(key == GLFW_KEY_A) {
        auto right = glm::cross(camera->up(), camera->direction());
        camera->set_position(camera->position() - right * speed);
    }
    else if(key == GLFW_KEY_Z) {
        camera->set_position(camera->position() + camera->up() * speed);
    }
    else if(key == GLFW_KEY_X) {
        camera->set_position(camera->position()  camera->up() * speed);
    }
    else if(key == GLFW_KEY_SPACE) {
        device_autofocus(camera,scene, WIDTH, HEIGHT);
    }
}
*/

void check_for_gl_errors() {
    while (true) {
        const GLenum err = glGetError();
        if (err == GL_NO_ERROR) {
            break;
        }

        std::cerr << "GL Error: " << gluErrorString(err) << std::endl;
    }
}

void create_gl_texture(GLuint *gl_tex, unsigned int size_x, unsigned int size_y) {
    glGenTextures(1, gl_tex);
    glBindTexture(GL_TEXTURE_2D, *gl_tex);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, size_x, size_y, 0, GL_RGBA, GL_FLOAT, NULL);

    check_for_gl_errors();
}

void init_glfw() {
    if (!glfwInit()) {
        std::cerr << "glfwInit failed!" << std::endl;
        exit(1);
    }
}

void init_gl_buffers() {
    create_gl_texture(&opengl_tex_cuda, WIDTH, HEIGHT);

    drawtex_v = GLSLShader("Textured draw vertex shader", glsl_drawtex_vertshader_src, GL_VERTEX_SHADER);
    drawtex_f = GLSLShader("Textured draw fragment shader", glsl_drawtex_fragshader_src, GL_FRAGMENT_SHADER);
    shdrawtex = GLSLProgram(&drawtex_v, &drawtex_f);
    shdrawtex.compile();
    check_for_gl_errors();
}

void display(Camera *camera, Scene *scene, Renderer &renderer, GlWindow &window, RandomGeneratorPool *random,
             size_t sample) {
    renderer.render(camera, scene, random, WIDTH, HEIGHT, sample);
    glfwPollEvents();


    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, opengl_tex_cuda);

    shdrawtex.use();
    glUniform1i(glGetUniformLocation(shdrawtex.program, "tex"), 0);

    window.draw();

    check_for_gl_errors();

    window.swap();
}

void print_cuda_device_info() {
    int device_count = 0;
    hipError_t error_id = hipGetDeviceCount(&device_count);

    std::cout << "Using the following CUDA device: " << std::endl;

    if (error_id != hipSuccess) {
        std::cerr << "hipGetDeviceCount returned " << (int) error_id << "\n" << hipGetErrorString(error_id)
                  << std::endl;
        exit(1);
    }

    if (device_count == 0) {
        std::cout << "There are no available devices that support CUDA" << std::endl;
        exit(1);
    }

    int device_id = 0;

    hipSetDevice(device_id);
    hipDeviceProp_t device_properties{};
    hipGetDeviceProperties(&device_properties, device_id);

    std::cout << "  Name: " << device_properties.name << "\n";

    int driver_version, runtime_version;
    hipDriverGetVersion(&driver_version);
    hipRuntimeGetVersion(&runtime_version);

    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driver_version / 1000,
           (driver_version % 100) / 10, runtime_version / 1000, (runtime_version % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n\n", device_properties.major,
           device_properties.minor);

}

std::vector<TriangleFace> faces_from_indices(const std::vector<int> &indices) {
    std::vector<TriangleFace> faces;
    for (int i = 0; i < indices.size(); i += 3) {
        faces.push_back({indices[i], indices[i + 1], indices[i + 2]});
    }

    return faces;
}

template<typename T, typename... Args>
T *create_device_type(Args &&... args) {
    T *object;
    hipMallocManaged(&object, sizeof(T));
    return new(object) T(std::forward<Args>(args)...);
}

void handle_input(GLFWwindow* window, Camera* camera, Scene *scene) {

    auto speed = 0.1f;
    if(glfwGetKey(window, GLFW_KEY_W)) {
        camera->set_position(camera->position() + camera->direction() * speed);
    }
    if(glfwGetKey(window, GLFW_KEY_S)) {
        camera->set_position(camera->position() - camera->direction() * speed);
    }
    if(glfwGetKey(window, GLFW_KEY_D)) {
        auto right = glm::cross(camera->up(), camera->direction());
        camera->set_position(camera->position() + right * speed);
    }
    if(glfwGetKey(window, GLFW_KEY_A)) {
        auto right = glm::cross(camera->up(), camera->direction());
        camera->set_position(camera->position() - right * speed);
    }
    if(glfwGetKey(window, GLFW_KEY_Z)) {
        camera->set_position(camera->position() + camera->up() * speed);
    }
    if(glfwGetKey(window, GLFW_KEY_X)) {
        camera->set_position(camera->position() - camera->up() * speed);
    }
     if(glfwGetKey(window, GLFW_KEY_SPACE)) {
        device_autofocus(camera,scene, WIDTH, HEIGHT);
    }
}

int main() {
    init_glfw();

    GlWindow window{"Hello, world!", WIDTH, HEIGHT, keyboard_func};

    init_gl_buffers();

    print_cuda_device_info();

    Renderer rend{opengl_tex_cuda, WIDTH, HEIGHT};

    auto camera = create_device_type<Camera>();

    float rot = 1.45f;
    //auto camera_position = glm::vec3(glm::cos(rot) * 10.0, 0.0000, glm::sin(rot) * 10.0f);
    auto camera_position = glm::vec3(2.0, 1.0, 7.5f);
    auto camera_direction = glm::normalize(glm::vec3(-7.0, 0.0, -60.0f) - camera_position);
    camera->set_position(camera_position);
    camera->set_direction(camera_direction);
    camera->set_up(glm::vec3(0.0, 1.0, 0.0));
    camera->set_field_of_view(90.0 * (3.1415 / 180.0));
    camera->set_blur_radius(0.0); // (0.03);
    camera->set_focal_length(60.0);
    camera->set_shutter_speed(0.0);
    camera->set_resolution(glm::vec2(WIDTH, HEIGHT));
    camera->update();

    DeviceMeshLoader mesh_loader;
    auto meshez = mesh_loader.load("/home/emil/models/house1/black_smith.obj"); // 0.5 0.35 0.5
    // auto meshez = mesh_loader.load("/home/emil/models/apple/apple.obj"); // 0.5 0.35 0.5
    // auto meshez = mesh_loader.load("/home/emil/models/crate/crate1.obj");
    // auto suzanne = meshez[0];

    //std::vector<IndexedDeviceMesh> meshes;
    //meshes.push_back(suzanne);

    auto grass = DeviceTextureLoader {}.load("/home/emil/textures/Grass004_4K-JPG/color.jpg");
    auto material = DeviceMaterial{grass};

    std::vector<glm::vec3> g_verts;
    g_verts.emplace_back(-1.0f, 0.0f, 1.0f);
    g_verts.emplace_back(1.0f, 0.0f, 1.0f);
    g_verts.emplace_back(1.0f, 0.0f, -1.0f);
    g_verts.emplace_back(-1.0f, 0.0f, -1.0f);
    g_verts.emplace_back(0.0f, -0.5f, 0.0f);

    /*g_verts.emplace_back(-1.0f, 0.1f, 1.0f);
    g_verts.emplace_back(1.0f, 0.1f, 1.0f);
    g_verts.emplace_back(1.0f, 0.1f, -1.0f);
    g_verts.emplace_back(-1.0f, 0.1f, -1.0f);*/

    std::vector<TriangleFace> g_faces;
    g_faces.push_back({0, 1, 2});
    g_faces.push_back({ 0, 2, 3});

    /*g_faces.push_back({4+0, 4+1, 4+2});
    g_faces.push_back({ 4+0, 4+2, 4+3});*/

    std::vector<glm::vec2> g_texcoords;
    g_texcoords.emplace_back(0.0f, 1.0f);
    g_texcoords.emplace_back(1.0f, 1.0f);
    g_texcoords.emplace_back(1.0f, 0.0f);
    g_texcoords.emplace_back(0.0f, 0.0f);

    /*g_texcoords.emplace_back(0.0f, 1.0f);
    g_texcoords.emplace_back(1.0f, 1.0f);
    g_texcoords.emplace_back(1.0f, 0.0f);
    g_texcoords.emplace_back(0.0f, 0.0f);*/

    auto plane = create_device_type<IndexedDeviceMesh>(g_verts, g_faces, g_texcoords, material);

    std::vector<SceneEntity> entities;


    entities.emplace_back(
            plane,
            WorldTransformBuilder()
                .with_translation({0.0, -3.3, 0.0})
                .with_scale({10.0, 1.0, 10.0})
                .build()
            );

    for(int i = 0; i < 10; ++i) {
        entities.emplace_back(meshez[0],
                              WorldTransformBuilder()
                                      .with_translation({-7.0, 0.0, i * -12.0})
                                      .with_rotation({0, 1.57, 0})
                                      .with_uniform_scale(1000.0f)
                                      .build());
    }


    Scene *scene;
    hipMallocManaged(&scene, sizeof(Scene));
    new(scene) Scene;
    scene->build(meshez, entities);

    auto random = create_device_type<RandomGeneratorPool>(2048, 123);

    double rotation = 0.0;
    double total_duration = 0.0f;
    double max_duration = 0.0f;
    int frame_counter = 0;
    size_t sample = 0;

    device_autofocus(camera, scene, WIDTH, HEIGHT);
    while (!window.should_close()) {
        handle_input(window.handle(), camera, scene);

        if(camera->needs_update()) {
            camera->update();
            sample = 0;
        }

        auto start = std::chrono::high_resolution_clock::now();
        display(camera, scene, rend, window, random, sample);
        ++sample;
        auto end = std::chrono::high_resolution_clock::now();
        auto frame_duration = std::chrono::duration<double, std::milli>(end - start);
        frame_counter++;
        if (frame_duration.count() > max_duration) {
            max_duration = frame_duration.count();
        }
        total_duration += frame_duration.count();
        std::cout << '\r' << "Frame time: " << frame_duration.count() << "ms\t\t Avg (10 frames): "
                  << (total_duration / frame_counter) << "ms\t\t Max: " << max_duration << "ms\tt Sample: "
                  << sample << "                    "
                  << std::flush;

        if (frame_counter == 10) {
            frame_counter = 0;
            total_duration = 0;
        }
        rotation += frame_duration.count() * 0.0005;

        check_for_gl_errors();
    }

    hipFree(camera);

    // https://stackoverflow.com/questions/14446495/cmake-project-structure-with-unit-tests
    // https://bitbucket.org/EmilNorden/physicstracer/src/master/CMakeLists.txt
    return 0;
}
