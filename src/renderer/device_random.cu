#include "hip/hip_runtime.h"
#include "device_random.cuh"
#include "hiprand/hiprand_kernel.h"

__global__ void init_random_states(hiprandState *states, size_t pool_size, unsigned long long seed) {
    for (auto i = 0; i < pool_size; ++i) {
        hiprand_init(seed, i, 0, &states[i]);
    }
}

void RandomGeneratorPool::init_global_state(size_t pool_size, unsigned long long seed) {
    init_random_states<<<1, 1>>>(m_global_state, pool_size, seed);
    hipDeviceSynchronize();
}