#include "hip/hip_runtime.h"
//
// Created by emil on 2021-05-09.
//

#include "renderer.cuh"

#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include <GL/glew.h>
#include <stdlib.h>
#include <stdio.h>
#include <cuda_gl_interop.h>

hipError_t cuda();

__global__ void kernel(){

}

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b) << 16) | (int(g) << 8) | int(r);
}

__global__ void
cudaRender(unsigned int *g_odata, int imgw, int offset)
{
    // extern __shared__ uchar4 sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx + offset;
    int y = blockIdx.y*bh + ty;

    uchar4 c4 = make_uchar4((x & 0x20) ? 100 : 0, 0, (y & 0x20) ? 100 : 0, 0);
    g_odata[y*imgw + x] = rgbToInt(c4.z, c4.y, c4.x);
}

void launch_cudaRender(dim3 grid, dim3 block, int sbytes, unsigned int *g_odata, int imgw, int offset)
{
    //cudaRender << < grid, block, sbytes >> >(g_odata, imgw, offset);
}

void Renderer::render(int width, int height) {
    dim3 block(16, 16, 1);
    dim3 grid(width / block.x, height / block.y, 1);
    cudaRender<<<grid, block, 0>>>((unsigned int*)m_cuda_render_buffer, width, 0);

    hipArray *texture_ptr;
    cuda_assert(hipGraphicsMapResources(1, &m_cuda_tex_resource, 0));
    cuda_assert(hipGraphicsSubResourceGetMappedArray(&texture_ptr, m_cuda_tex_resource, 0, 0));

    // TODO: Havent we already calculated this?
    int num_texels = width * height;
    int num_values = num_texels * 4;
    int size_tex_data = sizeof(GLubyte) * num_values;
    cuda_assert(hipMemcpyToArray(texture_ptr, 0, 0, m_cuda_render_buffer, size_tex_data, hipMemcpyDeviceToDevice));
    cuda_assert(hipGraphicsUnmapResources(1, &m_cuda_tex_resource, 0));
}

Renderer::Renderer(GLuint gl_texture, int width, int height)
    : m_cuda_render_buffer(nullptr) {
    allocate_render_buffer(width, height);

    cuda_assert(hipGraphicsGLRegisterImage(&m_cuda_tex_resource, gl_texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
}

void Renderer::allocate_render_buffer(int width, int height) {
    if(m_cuda_render_buffer) {
        hipFree(m_cuda_render_buffer);
    }

    auto buffer_size = width * height * 4 * sizeof(GLubyte); // Is GLubyte ever larger than 1?
    cuda_assert(hipMalloc(&m_cuda_render_buffer, buffer_size));
}

Renderer::~Renderer() {
    if(m_cuda_render_buffer) {
        hipFree(m_cuda_render_buffer);
    }
}
