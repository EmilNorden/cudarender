#include "hip/hip_runtime.h"
//
// Created by emil on 2021-05-09.
//

#include "coordinates.cuh"
#include "renderer.cuh"
#include "camera.cuh"

#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include <GL/glew.h>
#include <stdlib.h>
#include <stdio.h>
#include <cuda_gl_interop.h>

hipError_t cuda();

__global__ void kernel(){

}

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b) << 16) | (int(g) << 8) | int(r);
}

__device__ bool hit_sphere(const WorldSpaceRay& ray) {
    auto radius = 2.0f;
    auto position = glm::vec3(0.0, 0.0, 10.0);

    auto squared_radius = radius*radius;
    auto L = position - ray.origin().as_vec3();
    auto tca = glm::dot(L, ray.direction());

    auto d2 = glm::dot(L, L) - tca * tca;

    if(d2 > squared_radius) {
        return false;
    }

    auto thc = glm::sqrt(squared_radius - d2);
    auto t0 = tca - thc;
    auto t1 = tca + thc;

    if(t0 > t1) {
        auto temp = t0;
        t0 = t1;
        t1 = temp;
    }

    if(t0 < 0.0) {
        t0 = t1;
        if(t0 < 0.0) {
            return false;
        }
    }

    return true;
}


__global__ void
cudaRender(unsigned int *g_odata, Camera *camera, int width, int height)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

    /*Camera camera;
    camera.set_position(glm::vec3(0.0, 0.0, 0.0));
    camera.set_direction(glm::vec3(0.0, 0.0, 1.0));
    camera.set_up(glm::vec3(0.0, 1.0, 0.0));
    camera.set_field_of_view(90.0 * (3.1415 / 180.0));
    camera.set_blur_radius(0.0);
    camera.set_focal_length(1.0);
    camera.set_shutter_speed(0.0);
    camera.set_resolution(glm::vec2(width, height));

    camera.update();*/

    //uchar4 c4 = make_uchar4((x & 0x20) ? 100 : 0, 0, (y & 0x20) ? 100 : 0, 0);
    // g_odata[y*width + x] = rgbToInt(c4.z, c4.y, c4.x);

    if(x < width && y < height) {
        auto ray = camera->cast_ray(x, y);

        auto hit = hit_sphere(ray);

        auto factor_x = (x / (float)width);
        auto factor_y = (y / (float)height);
        if(hit) {
            g_odata[y*width + x] = rgbToInt(factor_x * 255, 0, factor_y * 255);
        }
        else {
            g_odata[y*width + x] = rgbToInt(0, 0, 0);
        }

    }

}




void Renderer::render(Camera* camera, int width, int height) {
    dim3 block(16, 16, 1);
    dim3 grid(width / block.x, std::ceil(height / (float)block.y), 1);
    cudaRender<<<grid, block, 0>>>((unsigned int*)m_cuda_render_buffer, camera, width, height);

    hipArray *texture_ptr;
    cuda_assert(hipGraphicsMapResources(1, &m_cuda_tex_resource, 0));
    cuda_assert(hipGraphicsSubResourceGetMappedArray(&texture_ptr, m_cuda_tex_resource, 0, 0));

    // TODO: Havent we already calculated this?
    int num_texels = width * height;
    int num_values = num_texels * 4;
    int size_tex_data = sizeof(GLubyte) * num_values;
    cuda_assert(hipMemcpyToArray(texture_ptr, 0, 0, m_cuda_render_buffer, size_tex_data, hipMemcpyDeviceToDevice));
    cuda_assert(hipGraphicsUnmapResources(1, &m_cuda_tex_resource, 0));
}

Renderer::Renderer(GLuint gl_texture, int width, int height)
    : m_cuda_render_buffer(nullptr) {
    allocate_render_buffer(width, height);

    cuda_assert(hipGraphicsGLRegisterImage(&m_cuda_tex_resource, gl_texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
}

void Renderer::allocate_render_buffer(int width, int height) {
    if(m_cuda_render_buffer) {
        hipFree(m_cuda_render_buffer);
    }

    auto buffer_size = width * height * 4 * sizeof(GLubyte); // Is GLubyte ever larger than 1?
    cuda_assert(hipMalloc(&m_cuda_render_buffer, buffer_size));
}

Renderer::~Renderer() {
    if(m_cuda_render_buffer) {
        hipFree(m_cuda_render_buffer);
    }
}

void Renderer::render(int width, int height, const Camera &camera, const Scene &scene)
{

}