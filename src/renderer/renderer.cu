#include "hip/hip_runtime.h"
//
// Created by emil on 2021-05-09.
//

#include "coordinates.cuh"
#include "renderer.cuh"
#include "camera.cuh"
#include "scene.cuh"

#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include <GL/glew.h>
#include <stdlib.h>
#include <stdio.h>
#include <cuda_gl_interop.h>
#include "device_random.cuh"

hipError_t cuda();

__global__ void kernel() {

}

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b) {
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b) {
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b) {
    r = clamp(r, 0.0f, 1.0f) * 255;
    g = clamp(g, 0.0f, 1.0f) * 255;
    b = clamp(b, 0.0f, 1.0f) * 255;
    return (int(255) << 24) | (int(b) << 16) | (int(g) << 8) | int(r);
}

// convert 8-bit integer to floating point rgb color
__device__ glm::vec3 int_to_rgb(int color) {
    auto r = static_cast<float>(color & 0xFF) / 255.0f;
    auto g = static_cast<float>((color & 0xFF00) >> 8) / 255.0f;
    auto b = static_cast<float>((color & 0xFF0000) >> 16) / 255.0f;

    return glm::vec3(r, g, b);
}

__device__ bool hit_sphere(const WorldSpaceRay &ray) {
    auto radius = 2.0f;
    auto position = glm::vec3(0.0, 0.0, 10.0);

    auto squared_radius = radius * radius;
    auto L = position - ray.origin();
    auto tca = glm::dot(L, ray.direction());

    auto d2 = glm::dot(L, L) - tca * tca;

    if (d2 > squared_radius) {
        return false;
    }

    auto thc = glm::sqrt(squared_radius - d2);
    auto t0 = tca - thc;
    auto t1 = tca + thc;

    if (t0 > t1) {
        auto temp = t0;
        t0 = t1;
        t1 = temp;
    }

    if (t0 < 0.0) {
        t0 = t1;
        if (t0 < 0.0) {
            return false;
        }
    }

    return true;
}


__global__ void
cudaRender(float *g_odata, Camera *camera, Scene *scene, RandomGeneratorPool *random_pool, int width, int height,
           size_t sample) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x * bw + tx;
    int y = blockIdx.y * bh + ty;

    auto threads_per_block = bw * bh;
    auto thread_num_in_block = tx + bw * ty;
    auto block_num_in_grid = blockIdx.x + gridDim.x * blockIdx.y;

    // auto global_thread_id = block_num_in_grid * threads_per_block + thread_num_in_block;
    auto global_block_id = block_num_in_grid;
    auto random = random_pool->get_generator(global_block_id);

    //uchar4 c4 = make_uchar4((x & 0x20) ? 100 : 0, 0, (y & 0x20) ? 100 : 0, 0);
    // g_odata[y*width + x] = rgbToInt(c4.z, c4.y, c4.x);

    if (x < width && y < height) {
        // auto ray = camera->cast_ray(x, y);
        auto ray = camera->cast_perturbed_ray(x, y, random);

        //auto hit = hit_sphere(ray);
        auto color = scene->hit(ray);

        glm::vec3 previous_color;

        auto pixel_index = y * (width*4) + (x*4);

        g_odata[pixel_index] = ((g_odata[pixel_index] * (float)sample) + color.x) / (sample + 1.0f);
        g_odata[pixel_index + 1] = ((g_odata[pixel_index + 1] * (float)sample) + color.y) / (sample + 1.0f);
        g_odata[pixel_index + 2] = ((g_odata[pixel_index + 2] * (float)sample) + color.z) / (sample + 1.0f);
        g_odata[pixel_index + 3] = 1.0;

        //auto previous_r = previous_color & 0x000000
//         g_odata[y*width + x] = ((previous_color * sample) + rgbToInt(color.x, color.y, color.z)) / (sample + 1);
    }

}

void Renderer::render(Camera *camera, Scene *scene, RandomGeneratorPool *random, int width, int height, size_t sample) {
    dim3 block(16, 16, 1);
    dim3 grid(width / block.x, std::ceil(height / (float) block.y), 1);
    cudaRender<<<grid, block, 0>>>((float*) m_cuda_render_buffer, camera, scene, random, width, height, sample);

    hipArray *texture_ptr;
    cuda_assert(hipGraphicsMapResources(1, &m_cuda_tex_resource, 0));
    cuda_assert(hipGraphicsSubResourceGetMappedArray(&texture_ptr, m_cuda_tex_resource, 0, 0));

    // TODO: Havent we already calculated this?
    int num_texels = width * height;
    int num_values = num_texels * 4;
    int size_tex_data = sizeof(GLfloat) * num_values;
    cuda_assert(hipMemcpyToArray(texture_ptr, 0, 0, m_cuda_render_buffer, size_tex_data, hipMemcpyDeviceToDevice));
    cuda_assert(hipGraphicsUnmapResources(1, &m_cuda_tex_resource, 0));
}

Renderer::Renderer(GLuint gl_texture, int width, int height)
        : m_cuda_render_buffer(nullptr) {
    allocate_render_buffer(width, height);

    cuda_assert(hipGraphicsGLRegisterImage(&m_cuda_tex_resource, gl_texture, GL_TEXTURE_2D,
                                            hipGraphicsRegisterFlagsWriteDiscard));
}

void Renderer::allocate_render_buffer(int width, int height) {
    if (m_cuda_render_buffer) {
        hipFree(m_cuda_render_buffer);
    }

    auto buffer_size = width * height * 4 * sizeof(GLfloat); // Is GLubyte ever larger than 1?
    cuda_assert(hipMalloc(&m_cuda_render_buffer, buffer_size));
}

Renderer::~Renderer() {
    if (m_cuda_render_buffer) {
        hipFree(m_cuda_render_buffer);
    }
}

void Renderer::render(int width, int height, const Camera &camera, const Scene &scene) {

}