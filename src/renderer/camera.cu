#include "hip/hip_runtime.h"
#include "camera.cuh"

#define PI 3.14159265359

__device__ WorldSpaceRay Camera::cast_ray(size_t x, size_t y) const
{
    // ray.set_origin(m_position);

    // glm::vec3 dir = (m_image_plane_origin - (m_u * m_pixel_width * (double)x) + (m_v * m_pixel_height * (double)y)) - m_position;
    // dir = glm::normalize(dir);
    // ray.set_direction(dir);
    auto direction =  (m_image_plane_origin - (m_u * m_pixel_width * (float)x) + (m_v * m_pixel_height * (float)y)) - m_position;
    direction = glm::normalize(direction);

    return WorldSpaceRay{m_position, direction};
    // ray.dist = DBL_MAX;
}

/*void Camera::cast_perturbed_ray(Ray &ray, size_t x, size_t y, std::mt19937 &rand) const
{
    cast_ray(ray, x, y);
    glm::vec3 focus_point = m_position + ray.m_direction * m_focal_length;

    std::uniform_real_distribution<float> distribution(0, 1);

    float angle = distribution(rand) * PI * 2.0f;
    float length = distribution(rand) * m_blur_radius;

    ray.m_origin = m_position + (m_u * std::sin(angle) * length) + (m_v * std::cos(angle) * length);
    ray.m_direction = glm::normalize(focus_point - ray.m_origin);
}*/

glm::vec2 Camera::project_onto_image_plane(const glm::vec3 &world_coord) const
{
    glm::vec3 coord_direction = glm::normalize(m_position - world_coord);

    float distance = glm::dot((m_image_plane_origin - m_position), m_direction) / glm::dot(coord_direction, m_direction);
    glm::vec3 image_plane_point = m_position + (distance * coord_direction);

    auto foo1 = m_image_plane_origin - image_plane_point;

    float x = glm::dot(m_u, foo1);
    float x1 = x / m_pixel_width;
    float y = glm::dot(m_v, foo1);
    float y1 = y / -m_pixel_height;

    return glm::vec2{x1, y1};
}