#include "device_texture_loader.cuh"

#include "device_texture.cuh"

#include <FreeImage.h>
#include <vector>
#include <filesystem>
#include <iostream>

using namespace std;

DeviceTexture *DeviceTextureLoader::load(const std::string &path) {
    cout << "Loading texture " << path << endl;
    if(!filesystem::exists(path)) {
        cerr << "  File does not exist!" << endl;
        exit(1);
    }

    FREE_IMAGE_FORMAT type = FreeImage_GetFileType(path.c_str());

    if (type == FIF_UNKNOWN) {
        type = FreeImage_GetFIFFromFilename(path.c_str());

        if (type == FIF_UNKNOWN){
            cerr << "  Unable to determine texture format!" << endl;
            exit(1);
        }
    }

    FIBITMAP *bitmap = FreeImage_Load(type, path.c_str());
    auto width = FreeImage_GetWidth(bitmap);
    auto height = FreeImage_GetHeight(bitmap);
    std::vector <uint8_t> pixels;
    pixels.reserve(3 * width * height);
    for (auto y = 0; y < height; ++y) {
        for (auto x = 0; x < width; ++x) {
            RGBQUAD pixel;
            FreeImage_GetPixelColor(bitmap, x, y, &pixel);
            pixels.push_back(pixel.rgbRed);
            pixels.push_back(pixel.rgbGreen);
            pixels.push_back(pixel.rgbBlue);
        }
    }

    FreeImage_Unload(bitmap);

    cout << endl;

    DeviceTexture *texture;
    hipMallocManaged(&texture, sizeof(DeviceTexture));
    return new(texture) DeviceTexture{pixels, width, height};
}

bool DeviceTextureLoader::file_is_supported(const std::string &path) {
    FREE_IMAGE_FORMAT type = FreeImage_GetFileType(path.c_str());

    if (type == FIF_UNKNOWN) {
        type = FreeImage_GetFIFFromFilename(path.c_str());
    }

    return type != FIF_UNKNOWN;
}