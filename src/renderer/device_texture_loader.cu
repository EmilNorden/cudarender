#include "device_texture_loader.cuh"

#include "device_texture.cuh"

#include <FreeImage.h>
#include <vector>
#include <filesystem>
#include <iostream>

using namespace std;

bool is_power_of_two(size_t n) {
    return (n & (n - 1)) == 0;
}

DeviceTexture *DeviceTextureLoader::load(const std::string &path) {
    cout << "Loading texture " << path << endl;
    if(!filesystem::exists(path)) {
        cerr << "  File does not exist!" << endl;
        exit(1);
    }

    FREE_IMAGE_FORMAT type = FreeImage_GetFileType(path.c_str());

    if (type == FIF_UNKNOWN) {
        type = FreeImage_GetFIFFromFilename(path.c_str());

        if (type == FIF_UNKNOWN){
            cerr << "  Unable to determine texture format!" << endl;
            exit(1);
        }
    }

    FIBITMAP *bitmap = FreeImage_Load(type, path.c_str());
    auto width = FreeImage_GetWidth(bitmap);
    auto height = FreeImage_GetHeight(bitmap);
    auto bpp = FreeImage_GetBPP(bitmap);

    if( !is_power_of_two(width) ||
        !is_power_of_two(height)) {
        cerr << "All textures must be power of 2!" << endl;
        exit(1);
    }

    if(bpp != 24) {
        auto new_bitmap = FreeImage_ConvertTo24Bits(bitmap);
        FreeImage_Unload(bitmap);
        bitmap = new_bitmap;
    }
    std::vector <uint8_t> pixels;
    pixels.reserve(3 * width * height);
    for (auto y = 0; y < height; ++y) {
        for (auto x = 0; x < width; ++x) {
            RGBQUAD pixel;
            FreeImage_GetPixelColor(bitmap, x, y, &pixel);
            pixels.push_back(pixel.rgbRed);
            pixels.push_back(pixel.rgbGreen);
            pixels.push_back(pixel.rgbBlue);
        }
    }

    FreeImage_Unload(bitmap);

    cout << endl;

    DeviceTexture *texture;
    hipMallocManaged(&texture, sizeof(DeviceTexture));
    return new(texture) DeviceTexture{pixels, width, height};
}

bool DeviceTextureLoader::file_is_supported(const std::string &path) {
    FREE_IMAGE_FORMAT type = FreeImage_GetFileType(path.c_str());

    if (type == FIF_UNKNOWN) {
        type = FreeImage_GetFIFFromFilename(path.c_str());
    }

    return type != FIF_UNKNOWN;
}