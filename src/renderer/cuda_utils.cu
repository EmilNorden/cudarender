//
// Created by emil on 2021-05-09.
//

#include "cuda_utils.cuh"
#include <iostream>

void cuda_assert(hipError_t err) {
    if(err != hipSuccess) {
        std::cerr << "CUDA call failed: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}
